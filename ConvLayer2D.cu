#include "ConvLayer2D.cuh"
#include <hipblas.h>
#include <random>
#include "Utils.cuh"

#define CUDA_CHECK(call) \
do { \
hipError_t err = call; \
if (err != hipSuccess) { \
    printf("CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
    exit(EXIT_FAILURE); \
} \
} while (0)

#define CUDNN_CHECK(call) \
do { \
hipdnnStatus_t status = call; \
if (status != HIPDNN_STATUS_SUCCESS) { \
    printf("cuDNN error in %s at line %d: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(status)); \
    exit(EXIT_FAILURE); \
} \
} while (0)


ConvLayer2D::ConvLayer2D(int batch, int in_channels, int in_height, int in_width,
    int out_channels, int kernel_size, int stride, int padding)
    : batch(batch), in_channels(in_channels), in_height(in_height), in_width(in_width),
    out_channels(out_channels), kernel_size(kernel_size), stride(stride), padding(padding) {

    CUDNN_CHECK(hipdnnCreate(&handle));

    // Input Descriptor
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&input_desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
        batch, in_channels, in_height, in_width));

    // Filter Descriptor
    CUDNN_CHECK(hipdnnCreateFilterDescriptor(&filter_desc));
    CUDNN_CHECK(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
        out_channels, in_channels, kernel_size, kernel_size));

    // Convolution Descriptor
    CUDNN_CHECK(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CHECK(hipdnnSetConvolution2dDescriptor(conv_desc, padding, padding, stride, stride, 1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    // Output Shape Calculation
    CUDNN_CHECK(hipdnnGetConvolution2dForwardOutputDim(conv_desc, input_desc, filter_desc,
        &batch, &out_channels, &out_height, &out_width));

    // Output Descriptor
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&output_desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
        batch, out_channels, out_height, out_width));

    // Bias Descriptor
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&bias_desc));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, out_channels, 1, 1));

    // Allocate Memory for filter, output, and bias
    CUDA_CHECK(hipMalloc(&d_filter, out_channels * in_channels * kernel_size * kernel_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_bias, out_channels * sizeof(float)));  // New: Bias
    CUDA_CHECK(hipMalloc(&d_output, batch * out_channels * out_height * out_width * sizeof(float)));

    // Initialize weights and bias (random init)
    std::vector<float> host_filter(out_channels * in_channels * kernel_size * kernel_size + out_channels);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::normal_distribution<float> dist(0.0f, 0.1f);

    for (auto& x : host_filter) x = dist(gen);

    CUDA_CHECK(hipMemcpy(d_filter, host_filter.data(), (host_filter.size() - out_channels) * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_bias, &host_filter.data()[host_filter.size() - out_channels],
        out_channels * sizeof(float), hipMemcpyHostToDevice));

    // Gradients
    CUDA_CHECK(hipMalloc(&d_input_grad, batch * in_channels * in_height * in_width * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_filter_grad, out_channels * in_channels * kernel_size * kernel_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_bias_grad, out_channels * sizeof(float)));
}


void ConvLayer2D::forward(float* d_input) {
    float alpha = 1.0f, beta = 0.0f;
    size_t workspace_size = 0;
    hipdnnConvolutionFwdAlgo_t algo;

    int returnedAlgoCount;
    hipdnnConvolutionFwdAlgoPerf_t perfResults;
    CUDNN_CHECK(hipdnnFindConvolutionForwardAlgorithm(handle, input_desc, filter_desc, conv_desc,
        output_desc, 1, &returnedAlgoCount, &perfResults));
    algo = perfResults.algo;

    CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(handle, input_desc, filter_desc, conv_desc,
        output_desc, algo, &workspace_size));
    void* d_workspace;
    CUDA_CHECK(hipMalloc(&d_workspace, workspace_size));

    CUDNN_CHECK(hipdnnConvolutionForward(handle, &alpha, input_desc, d_input, filter_desc, d_filter,
        conv_desc, algo, d_workspace, workspace_size, &beta, output_desc, d_output));

    CUDNN_CHECK(hipdnnAddTensor(handle, &alpha, bias_desc, d_bias, &alpha, output_desc, d_output));

    CUDA_CHECK(hipFree(d_workspace));
}

void ConvLayer2D::backwardData(float* d_input, float* d_output_grad) {
    float alpha = 1.0f, beta = 0.0f;
    hipdnnConvolutionBwdDataAlgo_t algo;
    size_t workspace_size = 0;

    int returnedAlgoCount;
    hipdnnConvolutionBwdDataAlgoPerf_t perfResults;
    hipdnnFindConvolutionBackwardDataAlgorithm(handle, filter_desc, output_desc, conv_desc,
        input_desc, 1, &returnedAlgoCount, &perfResults);
    algo = perfResults.algo;

    hipdnnGetConvolutionBackwardDataWorkspaceSize(handle, filter_desc, output_desc, conv_desc,
        input_desc, algo, &workspace_size);

    void* d_workspace;
    hipMalloc(&d_workspace, workspace_size);

    hipdnnConvolutionBackwardData(handle, &alpha, filter_desc, d_filter, output_desc,
        d_output_grad, conv_desc, algo, d_workspace, workspace_size,
        &beta, input_desc, d_input_grad);

    hipMemcpy(d_input, d_input_grad, batch * in_channels * in_height * in_width * sizeof(float), hipMemcpyDeviceToDevice);

    hipFree(d_workspace);
}

void ConvLayer2D::backwardFilter(float* d_input, float* d_output_grad) {
    float alpha = 1.0f, beta = 0.0f;
    hipdnnConvolutionBwdFilterAlgo_t algo;
    size_t workspace_size = 0;

    int returnedAlgoCount;
    hipdnnConvolutionBwdFilterAlgoPerf_t perfResults;
    hipdnnFindConvolutionBackwardFilterAlgorithm(handle, input_desc, output_desc, conv_desc,
        filter_desc, 1, &returnedAlgoCount, &perfResults);
    algo = perfResults.algo;

    hipdnnGetConvolutionBackwardFilterWorkspaceSize(handle, input_desc, output_desc, conv_desc,
        filter_desc, algo, &workspace_size);

    void* d_workspace;
    hipMalloc(&d_workspace, workspace_size);

    hipdnnConvolutionBackwardFilter(handle, &alpha, input_desc, d_input, output_desc,
        d_output_grad, conv_desc, algo, d_workspace, workspace_size,
        &beta, filter_desc, d_filter_grad);

    hipFree(d_workspace);
}

void ConvLayer2D::backwardBias(float* d_output_grad) {
    float alpha = 1.0f, beta = 0.0f;

    hipdnnTensorDescriptor_t bias_desc;
    hipdnnCreateTensorDescriptor(&bias_desc);
    hipdnnSetTensor4dDescriptor(bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, out_channels, 1, 1);

    hipdnnConvolutionBackwardBias(handle, &alpha, output_desc, d_output_grad,
        &beta, bias_desc, d_bias_grad);

    hipdnnDestroyTensorDescriptor(bias_desc);
}

// New: Apply gradients (SGD update for filter & bias)
void ConvLayer2D::updateWeights(float learning_rate) {
    float alpha = -learning_rate;

    int wgrad_size = out_channels * in_channels * kernel_size * kernel_size; // Number of gradients
    float clip_threshold = 5.0f; // Adjust as needed
    clipGradients << <(wgrad_size + 255) / 256, 256 >> > (d_filter_grad, wgrad_size, clip_threshold);

    int bgrad_size = out_channels;
    clipGradients << <(bgrad_size + 255) / 256, 256 >> > (d_bias_grad, bgrad_size, clip_threshold);

    hipDeviceSynchronize();

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    int filter_size = out_channels * in_channels * kernel_size * kernel_size;
    hipblasSaxpy(cublas_handle, filter_size, &alpha, d_filter_grad, 1, d_filter, 1);

    hipblasSaxpy(cublas_handle, out_channels, &alpha, d_bias_grad, 1, d_bias, 1);

    hipblasDestroy(cublas_handle);
}

void ConvLayer2D::backward(float* d_input, float* d_output_grad, float lr) {
    backwardData(d_input, d_output_grad);
    hipDeviceSynchronize();
    backwardFilter(d_input, d_output_grad);
    backwardBias(d_output_grad);
    updateWeights(lr);
}

float* ConvLayer2D::getOutput(int* outputSize) {
    if (outputSize)
    {
        *outputSize = batch * out_channels * out_height * out_width * sizeof(float);
    }
    return d_output;
}

float* ConvLayer2D::getInputGrad(int* inputGradSize) {
    if (inputGradSize)
    {
        *inputGradSize = batch * in_channels * in_height * in_width * sizeof(float);
    }
    return d_input_grad;
}


ConvLayer2D::~ConvLayer2D() {
    CUDA_CHECK(hipFree(d_filter));
    CUDA_CHECK(hipFree(d_bias));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_input_grad));
    CUDA_CHECK(hipFree(d_filter_grad));
    CUDA_CHECK(hipFree(d_bias_grad));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(input_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(bias_desc));
    CUDNN_CHECK(hipdnnDestroyTensorDescriptor(output_desc));
    CUDNN_CHECK(hipdnnDestroyFilterDescriptor(filter_desc));
    CUDNN_CHECK(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_CHECK(hipdnnDestroy(handle));
}
