#include "ConvLayer2D.cuh"
#include <hipblas.h>
#include <random>

ConvLayer2D::ConvLayer2D(int batch, int in_channels, int in_height, int in_width,
    int out_channels, int kernel_size, int stride, int padding)
    : batch(batch), in_channels(in_channels), in_height(in_height), in_width(in_width),
    out_channels(out_channels), kernel_size(kernel_size), stride(stride), padding(padding) {

    hipdnnCreate(&handle);

    // Input Descriptor
    hipdnnCreateTensorDescriptor(&input_desc);
    hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
        batch, in_channels, in_height, in_width);

    // Filter Descriptor
    hipdnnCreateFilterDescriptor(&filter_desc);
    hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
        out_channels, in_channels, kernel_size, kernel_size);

    // Convolution Descriptor
    hipdnnCreateConvolutionDescriptor(&conv_desc);
    hipdnnSetConvolution2dDescriptor(conv_desc, padding, padding, stride, stride, 1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

    // Output Shape Calculation
    hipdnnGetConvolution2dForwardOutputDim(conv_desc, input_desc, filter_desc,
        &batch, &out_channels, &out_height, &out_width);

    // Output Descriptor
    hipdnnCreateTensorDescriptor(&output_desc);
    hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
        batch, out_channels, out_height, out_width);

    // Bias Descriptor
    hipdnnCreateTensorDescriptor(&bias_desc);
    hipdnnSetTensor4dDescriptor(bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, out_channels, 1, 1);


    // Allocate Memory for filter, output, and bias
    hipMalloc(&d_filter, out_channels * in_channels * kernel_size * kernel_size * sizeof(float));
    hipMalloc(&d_bias, out_channels * sizeof(float));  // New: Bias
    hipMalloc(&d_output, batch * out_channels * out_height * out_width * sizeof(float));

    // Initialize weights and bias (optional: random init or zero)
    std::vector<float> host_filter(out_channels * in_channels * kernel_size * kernel_size + out_channels);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::normal_distribution<float> dist(0.0f, 0.1f);

    for (auto& x : host_filter) x = dist(gen);  // random init

    hipMemcpy(d_filter, host_filter.data(), (host_filter.size() - out_channels) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, &host_filter.data()[host_filter.size() - out_channels]
        , out_channels * sizeof(float), hipMemcpyHostToDevice);

    // Gradients
    hipMalloc(&d_input_grad, batch * in_channels * in_height * in_width * sizeof(float));
    hipMalloc(&d_filter_grad, out_channels * in_channels * kernel_size * kernel_size * sizeof(float));
    hipMalloc(&d_bias_grad, out_channels * sizeof(float));
}

void ConvLayer2D::forward(float* d_input) {
    float alpha = 1.0f, beta = 0.0f;
    size_t workspace_size = 0;
    hipdnnConvolutionFwdAlgo_t algo;

    // Select Fastest Algorithm
    int returnedAlgoCount;
    hipdnnConvolutionFwdAlgoPerf_t perfResults;
    hipdnnFindConvolutionForwardAlgorithm(handle, input_desc, filter_desc, conv_desc,
        output_desc, 1, &returnedAlgoCount, &perfResults);
    algo = perfResults.algo;

    hipdnnGetConvolutionForwardWorkspaceSize(handle, input_desc, filter_desc, conv_desc,
        output_desc, algo, &workspace_size);
    void* d_workspace;
    hipMalloc(&d_workspace, workspace_size);

    // Perform Convolution
    hipdnnConvolutionForward(handle, &alpha, input_desc, d_input, filter_desc, d_filter,
        conv_desc, algo, d_workspace, workspace_size, &beta,
        output_desc, d_output);

    hipdnnAddTensor(handle, &alpha, bias_desc, d_bias, &alpha, output_desc, d_output);

    hipFree(d_workspace);
}

void ConvLayer2D::backwardData(float* d_input, float* d_output_grad) {
    float alpha = 1.0f, beta = 0.0f;
    hipdnnConvolutionBwdDataAlgo_t algo;
    size_t workspace_size = 0;

    int returnedAlgoCount;
    hipdnnConvolutionBwdDataAlgoPerf_t perfResults;
    hipdnnFindConvolutionBackwardDataAlgorithm(handle, filter_desc, output_desc, conv_desc,
        input_desc, 1, &returnedAlgoCount, &perfResults);
    algo = perfResults.algo;

    hipdnnGetConvolutionBackwardDataWorkspaceSize(handle, filter_desc, output_desc, conv_desc,
        input_desc, algo, &workspace_size);

    void* d_workspace;
    hipMalloc(&d_workspace, workspace_size);

    hipdnnConvolutionBackwardData(handle, &alpha, filter_desc, d_filter, output_desc,
        d_output_grad, conv_desc, algo, d_workspace, workspace_size,
        &beta, input_desc, d_input_grad);

    hipMemcpy(d_input, d_input_grad, batch * in_channels * in_height * in_width * sizeof(float), hipMemcpyDeviceToDevice);

    hipFree(d_workspace);
}

void ConvLayer2D::backwardFilter(float* d_input, float* d_output_grad) {
    float alpha = 1.0f, beta = 0.0f;
    hipdnnConvolutionBwdFilterAlgo_t algo;
    size_t workspace_size = 0;

    int returnedAlgoCount;
    hipdnnConvolutionBwdFilterAlgoPerf_t perfResults;
    hipdnnFindConvolutionBackwardFilterAlgorithm(handle, input_desc, output_desc, conv_desc,
        filter_desc, 1, &returnedAlgoCount, &perfResults);
    algo = perfResults.algo;

    hipdnnGetConvolutionBackwardFilterWorkspaceSize(handle, input_desc, output_desc, conv_desc,
        filter_desc, algo, &workspace_size);

    void* d_workspace;
    hipMalloc(&d_workspace, workspace_size);

    hipdnnConvolutionBackwardFilter(handle, &alpha, input_desc, d_input, output_desc,
        d_output_grad, conv_desc, algo, d_workspace, workspace_size,
        &beta, filter_desc, d_filter_grad);

    hipFree(d_workspace);
}

void ConvLayer2D::backwardBias(float* d_output_grad) {
    float alpha = 1.0f, beta = 0.0f;

    hipdnnTensorDescriptor_t bias_desc;
    hipdnnCreateTensorDescriptor(&bias_desc);
    hipdnnSetTensor4dDescriptor(bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, out_channels, 1, 1);

    hipdnnConvolutionBackwardBias(handle, &alpha, output_desc, d_output_grad,
        &beta, bias_desc, d_bias_grad);

    hipdnnDestroyTensorDescriptor(bias_desc);
}

// New: Apply gradients (SGD update for filter & bias)
void ConvLayer2D::updateWeights(float learning_rate) {
    float alpha = -learning_rate;

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    int filter_size = out_channels * in_channels * kernel_size * kernel_size;
    hipblasSaxpy(cublas_handle, filter_size, &alpha, d_filter_grad, 1, d_filter, 1);

    hipblasSaxpy(cublas_handle, out_channels, &alpha, d_bias_grad, 1, d_bias, 1);

    hipblasDestroy(cublas_handle);
}

ConvLayer2D::~ConvLayer2D() {
    hipFree(d_filter);
    hipFree(d_bias);  // Free bias memory
    hipFree(d_output);
    hipFree(d_input_grad);
    hipFree(d_filter_grad);
    hipFree(d_bias_grad);
    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyTensorDescriptor(bias_desc);
    hipdnnDestroyTensorDescriptor(output_desc);
    hipdnnDestroyFilterDescriptor(filter_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroy(handle);
}
