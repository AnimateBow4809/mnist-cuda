#include "MaxPoolLayer.cuh"
#include <iostream>

#define CUDA_CHECK(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while (0)

#define CUDNN_CHECK(call) \
do { \
    hipdnnStatus_t status = call; \
    if (status != HIPDNN_STATUS_SUCCESS) { \
        std::cerr << "cuDNN error at " << __FILE__ << ":" << __LINE__ << " - " << hipdnnGetErrorString(status) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while (0)

MaxPoolLayer::MaxPoolLayer(int batch, int channels, int height, int width, int pool_size)
    : batch(batch), channels(channels), height(height), width(width), pool_size(pool_size) {

    pooled_height = (height - pool_size) / pool_size + 1;
    pooled_width = (width - pool_size) / pool_size + 1;

    CUDA_CHECK(hipMalloc(&d_output, batch * channels * pooled_height * pooled_width * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_input_grad, batch * channels * height * width * sizeof(float)));

    CUDNN_CHECK(hipdnnCreate(&cudnn));

    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&inputDesc));
    CUDNN_CHECK(hipdnnCreateTensorDescriptor(&outputDesc));
    CUDNN_CHECK(hipdnnCreatePoolingDescriptor(&poolingDesc));

    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch, channels, height, width));
    CUDNN_CHECK(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch, channels, pooled_height, pooled_width));

    int padding = 0;
    int stride = pool_size;
    CUDNN_CHECK(hipdnnSetPooling2dDescriptor(poolingDesc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN, pool_size, pool_size, padding, padding, stride, stride));
}

// Destructor
MaxPoolLayer::~MaxPoolLayer() {
    if (d_output) hipFree(d_output);
    if (d_input_grad) hipFree(d_input_grad);
    if (poolingDesc) hipdnnDestroyPoolingDescriptor(poolingDesc);
    if (inputDesc) hipdnnDestroyTensorDescriptor(inputDesc);
    if (outputDesc) hipdnnDestroyTensorDescriptor(outputDesc);
    if (cudnn) hipdnnDestroy(cudnn);
}

// Forward Pass
void MaxPoolLayer::forward(float* d_input) {
    float alpha = 1.0f, beta = 0.0f;
    CUDNN_CHECK(hipdnnPoolingForward(cudnn, poolingDesc, &alpha, inputDesc, d_input, &beta, outputDesc, d_output));
    hipDeviceSynchronize();
}

// Backward Pass
void MaxPoolLayer::backward(float* d_input, float* d_output_grad, float lr) {
    float alpha = 1.0f, beta = 0.0f;
    CUDA_CHECK(hipMemset(d_input_grad, 0, batch * channels * height * width * sizeof(float)));
    CUDNN_CHECK(hipdnnPoolingBackward(cudnn, poolingDesc, &alpha, outputDesc, d_output, outputDesc, d_output_grad,
        inputDesc, d_input, &beta, inputDesc, d_input_grad));
    hipDeviceSynchronize();

}

// Get Output
float* MaxPoolLayer::getOutput(int* outputSize) {
    if (outputSize) *outputSize = batch * channels * pooled_height * pooled_width * sizeof(float);
    return d_output;
}

// Get Input Gradient
float* MaxPoolLayer::getInputGrad(int* inputGradSize) {
    if (inputGradSize) *inputGradSize = batch * channels * height * width * sizeof(float);
    return d_input_grad;
}
