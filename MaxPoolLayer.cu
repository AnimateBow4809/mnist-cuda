#include "MaxPoolLayer.cuh"
#include <iostream>

MaxPoolLayer::MaxPoolLayer(int batch, int channels, int height, int width, int pool_size)
    : batch(batch), channels(channels), height(height), width(width), pool_size(pool_size) {

    pooled_height = height / pool_size;
    pooled_width = width / pool_size;

    hipMalloc(&d_output, batch * channels * pooled_height * pooled_width * sizeof(float));
    hipMalloc(&d_input_grad, batch * channels * height * width * sizeof(float));

    hipdnnCreate(&cudnn);

    // Define input and output tensor descriptors
    hipdnnCreateTensorDescriptor(&inputDesc);
    hipdnnCreateTensorDescriptor(&outputDesc);
    hipdnnCreatePoolingDescriptor(&poolingDesc);

    hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch, channels, height, width);
    hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch, channels, pooled_height, pooled_width);

    hipdnnSetPooling2dDescriptor(poolingDesc, HIPDNN_POOLING_MAX, HIPDNN_PROPAGATE_NAN, pool_size, pool_size, 0, 0, pool_size, pool_size);
}

// Destructor
MaxPoolLayer::~MaxPoolLayer() {
    hipFree(d_output);
    hipFree(d_input_grad);

    hipdnnDestroyPoolingDescriptor(poolingDesc);
    hipdnnDestroyTensorDescriptor(inputDesc);
    hipdnnDestroyTensorDescriptor(outputDesc);
    hipdnnDestroy(cudnn);
}

// Forward Pass
void MaxPoolLayer::forward(float* d_input) {
    float alpha = 1.0f, beta = 0.0f;
    hipdnnPoolingForward(cudnn, poolingDesc, &alpha, inputDesc, d_input, &beta, outputDesc, d_output);
}

// Backward Pass
void MaxPoolLayer::backward(float* d_input, float* d_output_grad, float lr) {
    float alpha = 1.0f, beta = 0.0f;
    hipdnnPoolingBackward(cudnn, poolingDesc, &alpha, outputDesc, d_output, outputDesc, d_output_grad,
        inputDesc, d_input, &beta, inputDesc, d_input_grad);
}

// Get Output
float* MaxPoolLayer::getOutput(int* outputSize) {
    if (outputSize) *outputSize = batch * channels * pooled_height * pooled_width;
    return d_output;
}

// Get Input Gradient
float* MaxPoolLayer::getInputGrad(int* inputGradSize) {
    if (inputGradSize) *inputGradSize = batch * channels * height * width;
    return d_input_grad;
}
