#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include "LossFunction.cuh";

#define CUDA_CHECK(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while (0)

__global__ void mse_forward_kernel(const float* pred, const float* target, float* loss, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int batch = blockIdx.y;

    __shared__ float cache[256];
    int tid = threadIdx.x;
    cache[tid] = 0.0f;

    if (idx < size) {
        float diff = pred[idx + batch * size] - target[idx + batch * size];
        cache[tid] = diff * diff;
    }
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            cache[tid] += cache[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(&loss[batch], cache[0]);
    }
}

__global__ void mse_backward_kernel(const float* pred, const float* target, float* grad, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int batch = blockIdx.y;

    if (idx < size) {
        grad[batch * size + idx] = 2.0f * (pred[batch * size + idx] - target[batch * size + idx]) / size;
    }
}

float* MSELoss::forward(const float* predictions, const float* targets, int size, int batch) {
    float* d_loss;
    CUDA_CHECK(hipMalloc(&d_loss, batch * sizeof(float)));
    CUDA_CHECK(hipMemset(d_loss, 0, batch * sizeof(float)));

    int threads = 256;
    dim3 blocks((size + threads - 1) / threads, batch);
    mse_forward_kernel << <blocks, threads >> > (predictions, targets, d_loss, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    float loss_host[200];
    CUDA_CHECK(hipMemcpy(loss_host, d_loss, batch * sizeof(float), hipMemcpyDeviceToHost));

    for (int i = 0; i < batch; i++) {
        loss_host[i] /= size;
    }

    CUDA_CHECK(hipMemcpy(d_loss, loss_host, batch * sizeof(float), hipMemcpyHostToDevice));
    return d_loss;
}

void MSELoss::backward(const float* predictions, const float* targets, float* grad, int size, int batch) {
    CUDA_CHECK(hipMemset(grad, 0, batch * size * sizeof(float)));
    int threads = 256;
    dim3 blocks((size + threads - 1) / threads, batch);
    mse_backward_kernel << <blocks, threads >> > (predictions, targets, grad, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}
