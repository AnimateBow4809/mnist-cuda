#include "hip/hip_runtime.h"
#include "ConvLayer2D.cuh"
#include <hipblas.h>
#include "LinearLayer.cuh"
#include <hip/hip_runtime.h>  // Core CUDA runtime API
#include <>  // Required for kernel launch parameters
#include <hiprand/hiprand_kernel.h>
#include "LossFunction.cuh"
#include "ReluLayer.cuh"
#include "MNISTTest.h"
#include"NNModel.cuh"
#include "DatasetLoader.cuh"
float* printGpuArray(float* d_in, int size, int newLine) {
    float* h_temp = (float*)malloc(size * sizeof(float));
    hipMemcpy(h_temp, d_in, size * sizeof(float), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < size; i++)
    {
        printf("%f ", h_temp[i]);
        if ((i + 1) % newLine == 0)
        {
            printf("\n");
        }
    }
    return h_temp;
}

float* initialiseGpuArrayRandom(float* d_in, int size,int newLine) {
    float* h_input = (float*)malloc(size* sizeof(float));
    srand(time(0));
    for (size_t i = 0; i < size; i++)
    {
        h_input[i] = (rand() / (float)RAND_MAX);
        if (newLine!=-1)
        {
            printf("%f ", h_input[i]);
            if ((i + 1) % newLine == 0)
            {
                printf("\n");
            }
        }
    }
    hipMemcpy(d_in, h_input, size* sizeof(float), hipMemcpyHostToDevice);
    return h_input;
}

void multiplyMatrix(float* d_matrix, int rows, int cols,float alpha, hipblasHandle_t handle) {
    int size = rows * cols;
    hipblasSscal(handle, size, &alpha, d_matrix, 1);
}






__global__ void matMulKernelRowMajor(float* A, float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; ++k) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

float* PushArrayIntoGpu(float* arr_h,dim3 dimentions) {
    float* arr_d;
    hipMalloc((void**)&arr_d, dimentions.x * dimentions.y * dimentions.z * sizeof(float));
    hipMemcpy(arr_d, arr_h, dimentions.x * dimentions.y * dimentions.z * sizeof(float), hipMemcpyHostToDevice);
    return arr_d;
}

float* createMatrix(int n, int m) {
    float* matrix = new float[n * m]; // Allocate memory for a 1D array
    for (int i = 0; i < n * m; ++i) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX; // Random value between 0 and 1
    }
    return matrix;
}

float* multMatrix(float* in,int row,int col, float alpha) {

    float* ans =(float*) malloc(sizeof(float) * row * col);
    for (size_t i = 0; i < row*col; i++)
    {
        ans[i] = alpha * in[i];
    }
    return ans;
}


int main() {
    float* train_images;
    float* train_labels;
    int num_train, img_size;



    read_mnist_images("t10k-images.idx3-ubyte", train_images, num_train, img_size);
    read_mnist_labels("train-labels.idx1-ubyte", train_labels, num_train);
    
    srand(static_cast<unsigned>(time(0))); // Seed for randomness
    hipblasHandle_t cchandle;
    hipblasCreate(&cchandle);
    //float arr_h[] = { 1,2,3,4,5,6,
    //            /*    7,8,9,10,11,12,
    //                13,14,15,16,17,18,
    //                19,20,21,22,23,24 
    //};
    int batch = 200;
    int input_feat = 28*28;
    int output_feat = 1;
    //int hidden = 10;
    dim3 dimensions_in(1, batch, input_feat);  // 1x4x6 tensor
    dim3 dimensions_out(1, batch, output_feat);  // 1x4x6 tensor
    
   // float* d_input = PushArrayIntoGpu(arr_h, dimensions_in);
    DatasetLoader image_loader(num_train/6, batch, 28, 28, train_images);
    DatasetLoader label_loader(num_train/6, batch, 1, 1, train_labels);

    std::vector<NNLayer*> layers;
    layers.push_back(new ConvLayer2D(batch, 1, 28, 28, 32, 3, 1, 1));
    layers.push_back(new ReLULayer(batch, 32, 28, 28));

    layers.push_back(new ConvLayer2D(batch, 32, 28, 28, 64, 3, 1, 1));
    layers.push_back(new ReLULayer(batch, 64, 28, 28));

    layers.push_back(new ConvLayer2D(batch, 64, 28, 28, 128, 3, 1, 1));
    layers.push_back(new ReLULayer(batch, 128, 28, 28));

    layers.push_back(new ConvLayer2D(batch, 128, 28, 28, 256, 3, 1, 1));
    layers.push_back(new ReLULayer(batch, 256, 28, 28));


    layers.push_back(new LinearLayer(batch, 256*28*28, 512));
    layers.push_back(new ReLULayer(batch, 1, 1, 512));

    layers.push_back(new LinearLayer(batch, 512, 1));

    NNModel model(layers);
    LossFunction* l1 = new MSELoss();

    hipDeviceSynchronize();
    float* d_grad;
    hipMalloc((void**) & d_grad, output_feat *batch*sizeof(float));

    for (int i = 0; i < 51; i++)
    {
        printf("\n%d iter:\n", i);
        float* target, * d_input;
        image_loader.Next(&d_input);
        label_loader.Next(&target);
        printf("Target:\n");
        printGpuArray(target, output_feat * batch, 10);
        model.forward(d_input);
        printf("\nResults:\n");
        float* h_output = printGpuArray(model.getOutput(), batch * output_feat, 10);

        float* d_loss = l1->forward(model.getOutput(), target, output_feat, batch);
        hipFree(d_loss);
        l1->backward(model.getOutput(), target, d_grad, output_feat, batch);
        float lr = 0.01;
        model.backward(d_input,d_grad,lr);
        hipDeviceSynchronize();
    }

    return 0;

}



//conv.forward(d_input);
//
//printf("bias:\n");
//float* h_bias = printGpuArray(conv.d_bias, 1, 1);
//
//printf("Filter:\n");
//float* h_filter = printGpuArray(conv.d_filter, 3 * 3, 3);
//
//printf("Results:\n");
//float* h_output = printGpuArray(conv.d_output, 3 * 3, 3);
//
//multiplyMatrix(conv.d_output, 3, 3, 2, handle);
////hipMemset(&conv.d_output[4], 0, 1 * sizeof(float));
//
//
//conv.backwardFilter(d_input, conv.d_output);
//conv.backwardBias(conv.d_output);
//
//conv.updateWeights(0.05);
//




////linear test
//// Define Conv Layer: (Batch=1, InChannels=1, Height=5, Width=5, OutChannels=1, Kernel=3, Stride=1, Padding=0)
    //ConvLayer conv(1, 1, 5, 5, 1, 3, 1, 0);
    //hipblasHandle_t handle;
    //hipblasCreate(&handle);
    //int input_feat = 3;
    //int output_feat = 10;

    //LinearLayer lin(1, input_feat, output_feat);

    //hipDeviceSynchronize();
    //int size = input_feat;

    //float *d_input,*d1_input;
    //hipMalloc((void**)&d_input, size*sizeof(float));
    //hipMalloc((void**)&d1_input, size*sizeof(float));

    //float* test = (float*)malloc(2 * sizeof(float));
    //test[0] = 0;
    //printf("input:\n");
    //float* h_input=initialiseGpuArrayRandom(d_input, input_feat,input_feat);
    //
    //for (size_t i = 0; i < 1; i++)
    //{

    //    lin.forward(d_input);
    //    printf("\nbias:\n");
    //    float* h_linbias = printGpuArray(lin.d_bias, output_feat, output_feat);
    //    printf("\nWeights:\n");
    //    float* h_linweights = printGpuArray(lin.d_weight, input_feat*output_feat, input_feat);
    //    printf("\nResults:\n");
    //    float* h_output = printGpuArray(lin.d_output, output_feat, output_feat);

    //    hipMemcpy(&test[1], &lin.d_output[2], 1 * sizeof(float), hipMemcpyDeviceToHost);
    //    test[0] = test[1] - 3.14; //output - target
    //    hipMemcpy(&lin.d_output[2], &test[0], 1 * sizeof(float), hipMemcpyHostToDevice);

    //    multiplyMatrix(lin.d_output, 1, output_feat, 2, handle);
    //    printf("\GRAD::\n");
    //    printGpuArray(lin.d_output, output_feat, output_feat);
    //    lin.backwardData(d_input, lin.d_output);
    //    lin.backwardWeights(d_input, lin.d_output);
    //    lin.backwardBias(lin.d_output);
    //    lin.updateWeights(0.05);

    //}


    //
    //return 0;



///// old loop

//for (size_t i = 0; i < 51; i++)
//{
//    printf("\n%d iter:\n", i);
//
//    //   float* h_input = createMatrix(batch,input_feat);
//    //   float* d_input = PushArrayIntoGpu(h_input, dimensions_in);
//      // float* h_target = multMatrix(h_input,batch,output_feat, 1000);
//      // float* target = PushArrayIntoGpu(h_target, dimensions_out);
//    float* target, * d_input;
//    image_loader.Next(&d_input);
//    label_loader.Next(&target);
//
//    // printf("Input:\n");
//     //printGpuArray(d_input, input_feat * batch, input_feat);
//
//    printf("Target:\n");
//    printGpuArray(target, output_feat * batch, 10);
//    model.forward(d_input);
//    //printf("\nbias:\n");
//    //float* h_linbias = printGpuArray(lin.d_bias, output_feat, output_feat);
//    //printf("\nWeights:\n");
//    //float* h_linweights = printGpuArray(lin.d_weight, input_feat*output_feat, input_feat);
//    printf("\nResults:\n");
//    float* h_output = printGpuArray(model.getOutput(), batch * output_feat, 10);
//    //float arr_h1[] = { 1,2,3,20,5,6,7 };
//    //            /* 1,2,3,4,5,6,7,
//    //             1,2,3,4,5,6,7,
//    //             1,2,3,4,5,6,7 };
//
//    //float* target = PushArrayIntoGpu(arr_h1, dimensions_out);
//
//    //printGpuArray(target, 7 * batch, 7);
//    float* d_loss = l1->forward(layers.at(layers.size() - 1)->getOutput(), target, output_feat, batch);
//    //printf("Loss:\n");
//    //printGpuArray(d_loss, batch, 1);
//    hipFree(d_loss);
//    l1->backward(layers.at(layers.size() - 1)->getOutput(), target, d_grad, output_feat, batch);
//    //printf("OUT_PUT_GRAD:\n");
//    //printGpuArray(d_grad, output_feat *batch, output_feat);
//    float lr = 0.01;
//    layers.at(layers.size() - 1)->backward(layers.at(layers.size() - 2)->getOutput(), d_grad, lr);
//    for (size_t i = layers.size() - 1; i > 0; i--)
//    {
//        layers.at(i)->backward(layers.at(i - 1)->getOutput(), layers.at(i - 1)->getInputGrad(), lr);
//    }
//    layers.at(0)->backward(d_input, layers.at(1)->getInputGrad(), lr);
//    hipDeviceSynchronize();
//}
//
//return 0;
