#include "hip/hip_runtime.h"
#include "ConvLayer.h"
#include <hipblas.h>
#include "LinearLayer.cuh"
#include <hip/hip_runtime.h>  // Core CUDA runtime API
#include <>  // Required for kernel launch parameters
#include <hiprand/hiprand_kernel.h>
#include "LossFunction.cuh";
#include "ReluLayer.cuh"
float* printGpuArray(float* d_in, int size, int newLine) {
    float* h_temp = (float*)malloc(size * sizeof(float));
    hipMemcpy(h_temp, d_in, size * sizeof(float), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < size; i++)
    {
        printf("%f ", h_temp[i]);
        if ((i + 1) % newLine == 0)
        {
            printf("\n");
        }
    }
    return h_temp;
}

float* initialiseGpuArrayRandom(float* d_in, int size,int newLine) {
    float* h_input = (float*)malloc(size* sizeof(float));
    srand(time(0));
    for (size_t i = 0; i < size; i++)
    {
        h_input[i] = (rand() / (float)RAND_MAX);
        if (newLine!=-1)
        {
            printf("%f ", h_input[i]);
            if ((i + 1) % newLine == 0)
            {
                printf("\n");
            }
        }
    }
    hipMemcpy(d_in, h_input, size* sizeof(float), hipMemcpyHostToDevice);
    return h_input;
}

void multiplyMatrix(float* d_matrix, int rows, int cols,float alpha, hipblasHandle_t handle) {
    int size = rows * cols;
    hipblasSscal(handle, size, &alpha, d_matrix, 1);
}






__global__ void matMulKernelRowMajor(float* A, float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; ++k) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

float* PushArrayIntoGpu(float* arr_h,dim3 dimentions) {
    float* arr_d;
    hipMalloc((void**)&arr_d, dimentions.x * dimentions.y * dimentions.z * sizeof(float));
    hipMemcpy(arr_d, arr_h, dimentions.x * dimentions.y * dimentions.z * sizeof(float), hipMemcpyHostToDevice);
    return arr_d;
}

float* createMatrix(int n, int m) {
    float* matrix = new float[n * m]; // Allocate memory for a 1D array
    for (int i = 0; i < n * m; ++i) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX; // Random value between 0 and 1
    }
    return matrix;
}

float* multMatrix(float* in,int row,int col, float alpha) {

    float* ans =(float*) malloc(sizeof(float) * row * col);
    for (size_t i = 0; i < row*col; i++)
    {
        ans[i] = alpha * in[i];
    }
    return ans;
}



int main() {
    srand(static_cast<unsigned>(time(0))); // Seed for randomness
    hipblasHandle_t cchandle;
    hipblasCreate(&cchandle);
    //float arr_h[] = { 1,2,3,4,5,6,
    //            /*    7,8,9,10,11,12,
    //                13,14,15,16,17,18,
    //                19,20,21,22,23,24 */
    //};
    int batch = 5;
    int input_feat = 6;
    int output_feat = 6;
    int hidden = 10;
    dim3 dimensions_in(1, batch, input_feat);  // 1x4x6 tensor
    dim3 dimensions_out(1, batch, output_feat);  // 1x4x6 tensor

   // float* d_input = PushArrayIntoGpu(arr_h, dimensions_in);

    

    LinearLayer lin(batch, input_feat, hidden);
    ReLULayer rel(batch, 1, 1, hidden);
    LinearLayer lin2(batch, hidden, output_feat);

    LossFunction* l1 = new MSELoss();

    hipDeviceSynchronize();
    float* d_grad;
    hipMalloc((void**) & d_grad, output_feat *batch*sizeof(float));

    for (size_t i = 0; i < 100; i++)
    {
        printf("\n%d iter:\n", i);

        float* h_input = createMatrix(batch,input_feat);
        float* d_input = PushArrayIntoGpu(h_input, dimensions_in);

        float* h_target = multMatrix(h_input,batch,output_feat, 1000);
        float* target = PushArrayIntoGpu(h_target, dimensions_out);

        printf("Input:\n");
        printGpuArray(d_input, input_feat * batch, input_feat);

        printf("Target:\n");
        printGpuArray(target, output_feat * batch, output_feat);



        lin.forward(d_input);
        rel.forward(lin.d_output);
        lin2.forward(rel.d_output);


        //printf("\nbias:\n");
        //float* h_linbias = printGpuArray(lin.d_bias, output_feat, output_feat);
        //printf("\nWeights:\n");
        //float* h_linweights = printGpuArray(lin.d_weight, input_feat*output_feat, input_feat);
        printf("\nResults:\n");
        float* h_output = printGpuArray(lin2.d_output, batch*output_feat, output_feat);
        //float arr_h1[] = { 1,2,3,20,5,6,7 };
        //            /* 1,2,3,4,5,6,7,
        //             1,2,3,4,5,6,7,
        //             1,2,3,4,5,6,7 };*/


        //float* target = PushArrayIntoGpu(arr_h1, dimensions_out);

        //printGpuArray(target, 7 * batch, 7);
        float* d_loss = l1->forward(lin2.d_output, target, output_feat, batch);
        printf("Loss:\n");
        //printGpuArray(d_loss, batch, 1);
        hipFree(d_loss);
        l1->backward(lin2.d_output, target, d_grad, output_feat, batch);
        printf("OUT_PUT_GRAD:\n");
        printGpuArray(d_grad, output_feat *batch, output_feat);

        lin2.backwardData(rel.d_output, d_grad);
        lin2.backwardWeights(rel.d_output, d_grad);
        lin2.backwardBias(d_grad);

       // hipDeviceSynchronize();
        
        rel.backward(lin.d_output, lin2.d_input_grad);

        lin.backwardData(d_input, rel.d_input_grad);
        lin.backwardWeights(d_input, rel.d_input_grad);
        lin.backwardBias(rel.d_input_grad);
        lin.updateWeights(0.01);
        lin2.updateWeights(0.01);
        hipDeviceSynchronize();
       

    }

    return 0;

}



//conv.forward(d_input);
//
//printf("bias:\n");
//float* h_bias = printGpuArray(conv.d_bias, 1, 1);
//
//printf("Filter:\n");
//float* h_filter = printGpuArray(conv.d_filter, 3 * 3, 3);
//
//printf("Results:\n");
//float* h_output = printGpuArray(conv.d_output, 3 * 3, 3);
//
//multiplyMatrix(conv.d_output, 3, 3, 2, handle);
////hipMemset(&conv.d_output[4], 0, 1 * sizeof(float));
//
//
//conv.backwardFilter(d_input, conv.d_output);
//conv.backwardBias(conv.d_output);
//
//conv.updateWeights(0.05);
//




////linear test
//// Define Conv Layer: (Batch=1, InChannels=1, Height=5, Width=5, OutChannels=1, Kernel=3, Stride=1, Padding=0)
    //ConvLayer conv(1, 1, 5, 5, 1, 3, 1, 0);
    //hipblasHandle_t handle;
    //hipblasCreate(&handle);
    //int input_feat = 3;
    //int output_feat = 10;

    //LinearLayer lin(1, input_feat, output_feat);

    //hipDeviceSynchronize();
    //int size = input_feat;

    //float *d_input,*d1_input;
    //hipMalloc((void**)&d_input, size*sizeof(float));
    //hipMalloc((void**)&d1_input, size*sizeof(float));

    //float* test = (float*)malloc(2 * sizeof(float));
    //test[0] = 0;
    //printf("input:\n");
    //float* h_input=initialiseGpuArrayRandom(d_input, input_feat,input_feat);
    //
    //for (size_t i = 0; i < 1; i++)
    //{

    //    lin.forward(d_input);
    //    printf("\nbias:\n");
    //    float* h_linbias = printGpuArray(lin.d_bias, output_feat, output_feat);
    //    printf("\nWeights:\n");
    //    float* h_linweights = printGpuArray(lin.d_weight, input_feat*output_feat, input_feat);
    //    printf("\nResults:\n");
    //    float* h_output = printGpuArray(lin.d_output, output_feat, output_feat);

    //    hipMemcpy(&test[1], &lin.d_output[2], 1 * sizeof(float), hipMemcpyDeviceToHost);
    //    test[0] = test[1] - 3.14; //output - target
    //    hipMemcpy(&lin.d_output[2], &test[0], 1 * sizeof(float), hipMemcpyHostToDevice);

    //    multiplyMatrix(lin.d_output, 1, output_feat, 2, handle);
    //    printf("\GRAD::\n");
    //    printGpuArray(lin.d_output, output_feat, output_feat);
    //    lin.backwardData(d_input, lin.d_output);
    //    lin.backwardWeights(d_input, lin.d_output);
    //    lin.backwardBias(lin.d_output);
    //    lin.updateWeights(0.05);

    //}


    //
    //return 0;